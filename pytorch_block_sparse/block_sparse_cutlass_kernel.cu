#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <string>
#include <fstream>
#include <sstream>

// CUBLAS GEMM API
#include <hipblas.h>

// Set Cutlass debug macro to enable console printing of library errors
#define DEBUG

// Cutlass GEMM API
#include <cutlass/util/util.h>
#include <cutlass/gemm/dispatch.h>
#include <cutlass/gemm/epilogue_function.h>

// Test utilities
#include "util/command_line.h"
#include "util/matrix.h"
#include "util/timer.h"
#include "util/type_conversion.h"


// Dispatch routines to CUTLASS
#include "cutlass_dispatch.h"

using namespace std;
using namespace cutlass;


hipblasHandle_t g_cublas_handle;
bool cublas_inited = false;

/**
 * Compute C = (alpha * A * B) + (beta * C)
 */
template <
    typename                        test_func_t,    ///< Test function type
    gemm::tiling_strategy::kind_t   TilingStrategy,
    matrix_transform_t::kind_t      TransformA,     ///< Transformation op for matrix A
    matrix_transform_t::kind_t      TransformB,     ///< Transformation op for matrix B
    typename                        value_t,        ///< Multiplicand value type (matrices A and B)
    typename                        accum_t>        ///< Accumulator value type (matrix C and scalars)
bool test_bsc(
    value_t* A_data,
    value_t* B_data,
    int* B_bsc_ptr,
    int* B_bsc_indices,
    accum_t* C_data,
    int m,          ///< Height of C in rows
    int n,          ///< Width of C in columns
    int k,          ///< Width (height) of A (B)
    accum_t alpha,  ///< Multiplicand scalar
    accum_t beta)
{

    typedef gemm::gemm_policy<value_t, accum_t, TransformA, TransformB, TilingStrategy> block_task_policy_t;

    // matrix pruning
    int BlockItemsN = block_task_policy_t::BlockItemsX; // depend on the block task policy
    int BlockItemsK = block_task_policy_t::BlockItemsK;

    printf("BlockItemsN=%d, BlockItemsK=%d", BlockItemsN, BlockItemsK);

    hipStream_t stream = 0;

    test_func_t test_func;

    hipError_t error = test_func(
        g_cublas_handle,
        m,
        n,
        k,
        A_data,
        B_data,
        B_bsc_ptr,
        B_bsc_indices,
        C_data,
        alpha,
        beta,
        stream,
        false).result;

    return error;
}

torch::Tensor  blocksparse_matmul_cutlass(torch::Tensor dense_a,
								      torch::Tensor row_start_ends_a,
								      torch::Tensor cols_a_0,
								      torch::Tensor data_b,
								      int m,
								      int n,
								      int k,
								      int block_size_rows_b,
								      int block_size_cols_b,
								      torch::Tensor dense_out)
{
    typedef float       value_t;
	typedef float       accum_t;
	const math_operation_class_t math_op = math_operation_class_t::scalar;
    static const matrix_transform_t::kind_t TransformA = matrix_transform_t::NonTranspose;
    static const matrix_transform_t::kind_t TransformB = matrix_transform_t::NonTranspose;

    value_t* A_data = (value_t*)dense_a.data_ptr();
    value_t* B_data = (value_t*)data_b.data_ptr();
    int* B_bsc_ptr = (int*)row_start_ends_a.data_ptr();
    int* B_bsc_indices = (int*)cols_a_0.data_ptr();
    value_t* C_data = (value_t*)dense_out.data_ptr();

    //int m = sizes_a[0];

    float alpha = 1.0;
    float beta = 0.0;

// Initialize cuBLAS
	if (!cublas_inited) {
		if (hipblasCreate(&g_cublas_handle) != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "hipblasCreate() failed\n");
			exit(1);
		}
		cublas_inited = true;
	}

	bool test_error = test_bsc<
	cutlass_gemm_dispatch<gemm::tiling_strategy::Custom, math_op, TransformA, TransformB, value_t, accum_t>,
	gemm::tiling_strategy::Custom,
	TransformA,
	TransformB,
	value_t,
	accum_t>(A_data,B_data,B_bsc_ptr, B_bsc_indices, C_data, m, n, k, accum_t(alpha), accum_t(beta));

    return dense_out;
}
