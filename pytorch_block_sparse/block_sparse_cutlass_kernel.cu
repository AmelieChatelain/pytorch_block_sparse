#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
#include <iostream>
#include <typeinfo>
#include <stdint.h>
#include <string>
#include <fstream>
#include <sstream>
*/

// CUBLAS GEMM API
#include <hipblas.h>

// Cutlass GEMM API
#include <cutlass/util/util.h>
#include <cutlass/gemm/dispatch.h>
#include <cutlass/gemm/epilogue_function.h>

// Dispatch routines to CUTLASS
#include "cutlass_dispatch.h"

using namespace std;
using namespace cutlass;

/**
 * Compute C = (alpha * A * B) + (beta * C), where B is block sparse, A and B dense
 **/
template <
    typename                        func_t,    ///< Test function type
    gemm::tiling_strategy::kind_t   TilingStrategy,
    matrix_transform_t::kind_t      TransformA,     ///< Transformation op for matrix A
    matrix_transform_t::kind_t      TransformB,     ///< Transformation op for matrix B
    typename                        value_t,        ///< Multiplicand value type (matrices A and B)
    typename                        accum_t>        ///< Accumulator value type (matrix C and scalars)
hipError_t forward(
    value_t* A_data,
    value_t* B_data,
    int* B_bsc_ptr,
    int* B_bsc_indices,
    accum_t* C_data,
    int m,          ///< Height of C in rows
    int n,          ///< Width of C in columns
    int k,          ///< Width (height) of A (B)
    accum_t alpha,  ///< Multiplicand scalar
    accum_t beta)
{

    typedef gemm::gemm_policy<value_t, accum_t, TransformA, TransformB, TilingStrategy> block_task_policy_t;

    // matrix pruning
    int BlockItemsN = block_task_policy_t::BlockItemsX; // depend on the block task policy
    int BlockItemsK = block_task_policy_t::BlockItemsK;

    hipStream_t stream = 0;

    func_t func;

    hipError_t error = func(
        m,
        n,
        k,
        A_data,
        B_data,
        B_bsc_ptr,
        B_bsc_indices,
        C_data,
        alpha,
        beta,
        stream,
        false).result;

    return error;
}

int blocksparse_matmul_cutlass(torch::Tensor dense_a,
  							    torch::Tensor ptr_b,
							    torch::Tensor indices_b,
							    torch::Tensor data_b,
							    int m,
							    int n,
							    int k,
							    int block_size_rows_b,
							    int block_size_cols_b,
							    torch::Tensor dense_out)
{
    typedef float       value_t;
	typedef float       accum_t;
	const math_operation_class_t math_op = math_operation_class_t::scalar;
    static const matrix_transform_t::kind_t TransformA = matrix_transform_t::Transpose;
    static const matrix_transform_t::kind_t TransformB = matrix_transform_t::Transpose;

    value_t* A_data = (value_t*)dense_a.data_ptr();
    value_t* B_data = (value_t*)data_b.data_ptr();
    int* B_ptr = (int*)ptr_b.data_ptr();
    int* B_indices = (int*)indices_b.data_ptr();
    value_t* C_data = (value_t*)dense_out.data_ptr();

    float alpha = 1.0;
    float beta = 0.0;

	hipError_t error = forward<cutlass_gemm_dispatch<gemm::tiling_strategy::Custom,
	                                                 math_op,
	                                                 TransformA,
	                                                 TransformB,
	                                                 value_t,
	                                                 accum_t>,
						       gemm::tiling_strategy::Custom,
					           TransformA,
						       TransformB,
						       value_t,
						       accum_t>(A_data,B_data,B_ptr, B_indices, C_data, m, n, k, accum_t(alpha), accum_t(beta));
    return error;
}


