#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <string>
#include <fstream>
#include <sstream>

// CUBLAS GEMM API
#include <hipblas.h>

// Set Cutlass debug macro to enable console printing of library errors
#define DEBUG

// Cutlass GEMM API
#include <cutlass/util/util.h>
#include <cutlass/gemm/dispatch_back.h>
#include <cutlass/gemm/epilogue_function.h>

// Test utilities
#include "util/command_line.h"
#include "util/matrix.h"
#include "util/timer.h"
#include "util/type_conversion.h"


// Dispatch routines to CUTLASS
#include "cutlass_dispatch_back.h"

using namespace std;
using namespace cutlass;


extern hipblasHandle_t g_cublas_handle;
extern bool cublas_inited;

/**
 * Compute C = (alpha * A * B) + (beta * C)
 */
template <
    typename                        test_func_t,    ///< Test function type
    gemm::tiling_strategy::kind_t   TilingStrategy,
    matrix_transform_t::kind_t      TransformA,     ///< Transformation op for matrix A
    matrix_transform_t::kind_t      TransformB,     ///< Transformation op for matrix B
    typename                        value_t,        ///< Multiplicand value type (matrices A and B)
    typename                        accum_t>        ///< Accumulator value type (matrix C and scalars)
bool test_bsc_back(
    value_t* A_data,
    value_t* B_data,
    accum_t* C_data,
    int2* C_blocks,
    long C_blocks_length,
    int m,          ///< Height of C in rows
    int n,          ///< Width of C in columns
    int k,          ///< Width (height) of A (B)
    accum_t alpha,  ///< Multiplicand scalar
    accum_t beta)
{

    typedef gemm::gemm_policy<value_t, accum_t, TransformA, TransformB, TilingStrategy> block_task_back_policy_t;

    // matrix pruning
    int BlockItemsN = block_task_back_policy_t::BlockItemsX; // depend on the block task policy
    int BlockItemsK = block_task_back_policy_t::BlockItemsK;

    hipStream_t stream = 0;

    test_func_t test_func;

    hipError_t error = test_func(
        g_cublas_handle,
        m,
        n,
        k,
        A_data,
        B_data,
        C_data,
        C_blocks,
        C_blocks_length,
        alpha,
        beta,
        stream,
        false).result;

    return error;
}

torch::Tensor  blocksparse_matmul_back_cutlass(torch::Tensor dense_a,
											  torch::Tensor dense_b,
											  int m,
											  int n,
											  int k,
											  int block_size_rows_b,
											  int block_size_cols_b,
											  torch::Tensor sparse_c,
											  torch::Tensor sparse_c_blocks,
											  long sparse_c_blocks_length
											  )
{
    typedef float       value_t;
	typedef float       accum_t;
	const math_operation_class_t math_op = math_operation_class_t::scalar;
    static const matrix_transform_t::kind_t TransformA = matrix_transform_t::Transpose;
    static const matrix_transform_t::kind_t TransformB = matrix_transform_t::Transpose;

    value_t* A_data = (value_t*)dense_a.data_ptr();
    value_t* B_data = (value_t*)dense_b.data_ptr();
    value_t* C_data = (value_t*)sparse_c.data_ptr();
    int2* C_blocks = (int2*)sparse_c_blocks.data_ptr();
    long C_blocks_length = sparse_c_blocks_length;

    //int m = sizes_a[0];

    float alpha = 1.0;
    float beta = 0.0;

// Initialize cuBLAS
	if (!cublas_inited) {
		if (hipblasCreate(&g_cublas_handle) != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "hipblasCreate() failed\n");
			exit(1);
		}
		cublas_inited = true;
	}

	bool test_error = test_bsc_back<
	cutlass_gemm_dispatch_back<gemm::tiling_strategy::CustomBack, math_op, TransformA, TransformB, value_t, accum_t>,
	gemm::tiling_strategy::CustomBack,
	TransformA,
	TransformB,
	value_t,
	accum_t>(A_data,B_data, C_data, C_blocks, C_blocks_length, m, n, k, accum_t(alpha), accum_t(beta));

    return sparse_c;
}
